
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <time.h>


#define GO_EMPTY 0
#define GO_BLACK 1
#define GO_WHITE 2
#define GO_BORDER 3

 
const int boardSize = 21; 
const int totalSize = boardSize * boardSize;


struct BoardPoint{
  int color;
  int groupID;
  int libertyNumber;
  bool isBlackLegal;
  bool isWhiteLegal;
};

struct DebugFlag{
  int counter;
  int changeFlag;
  int targetGroupID[4];
  int libertyCount;
};
 
__global__
void initBoard(BoardPoint *boardDevice){
 
  int index = threadIdx.y * boardSize + threadIdx.x;

  if (threadIdx.x == 0 || threadIdx.x == boardSize-1 || threadIdx.y == 0 || threadIdx.y == boardSize-1){
    boardDevice[index].color = 3;
  } else {
    boardDevice[index].color = 0;
  }

  //boardDevice[index].groupID = totalSize; // all the initial group ID was set to none group id.

}

__device__

inline void updateLiberty(BoardPoint *boardDevice, int index, int *globalLiberty){
   if (boardDevice[index].color == GO_EMPTY){

    atomicAdd(&globalLiberty[boardDevice[index-1].groupID], 1);
  
    if (boardDevice[index+boardSize].groupID != boardDevice[index-1].groupID){
      atomicAdd(&globalLiberty[boardDevice[index+boardSize].groupID], 1);
    } 

    if (boardDevice[index+1].groupID != boardDevice[index-1].groupID &&
        boardDevice[index+1].groupID != boardDevice[index+boardSize].groupID){
      atomicAdd(&globalLiberty[boardDevice[index+1].groupID], 1);
    } 

    if (boardDevice[index-boardSize].groupID != boardDevice[index-1].groupID &&
        boardDevice[index-boardSize].groupID != boardDevice[index+1].groupID &&
        boardDevice[index-boardSize].groupID != boardDevice[index+boardSize].groupID){
      atomicAdd(&globalLiberty[boardDevice[index-boardSize].groupID], 1);
    } 

  }
}

__global__
void playBoard(BoardPoint *boardDevice, DebugFlag *debugFlagDevice, int row, int col, int color){
  int index = threadIdx.y*boardSize + threadIdx.x;
  int playPoint = row*boardSize + col;

  __shared__ int globalLiberty[totalSize]; // shared array to count the liberty of each group.
  __shared__ int targetGroupID[4] ;
  __shared__ bool hasStoneRemoved;


  if (threadIdx.y == 0 || threadIdx.y == boardSize || threadIdx.x == 0 || threadIdx.x == boardSize){
    globalLiberty[0] = 0;
    return;
  }


  if (index == playPoint){
      boardDevice[index].color = color;
      boardDevice[index].groupID = index;

      if (boardDevice[index+1].color == color){
        targetGroupID[0] = boardDevice[index+1].groupID;
      }else{
        targetGroupID[0] = -1;
      }

      if (boardDevice[index-1].color == color){
        targetGroupID[1] = boardDevice[index-1].groupID;
      }else{
        targetGroupID[1] = -1;
      }
      
      if (boardDevice[index+boardSize].color == color){
        targetGroupID[2] = boardDevice[index+boardSize].groupID;
      }else{
        targetGroupID[2] = -1;
      }

      if (boardDevice[index-boardSize].color == color){
        targetGroupID[3] = boardDevice[index-boardSize].groupID;
      }else{
        targetGroupID[3] = -1;
      }

  }

  __syncthreads();

  //@todo , check whether this fence is necessory.
  __threadfence_block();


  if (boardDevice[index].groupID == targetGroupID[0] ||
      boardDevice[index].groupID == targetGroupID[1] ||
      boardDevice[index].groupID == targetGroupID[2] ||
      boardDevice[index].groupID == targetGroupID[3] ){
    boardDevice[index].groupID = playPoint;
  }

  globalLiberty[index] = 0;
  hasStoneRemoved = false;

  __syncthreads();
  __threadfence_block();

  updateLiberty(boardDevice, index, globalLiberty);

  __syncthreads();
  __threadfence_block();

  int libertyNumber = globalLiberty[boardDevice[index].groupID];
  if ( libertyNumber == 0 ){
    boardDevice[index].color = GO_EMPTY;
    boardDevice[index].groupID = 0;
    boardDevice[index].libertyNumber = 0;
    hasStoneRemoved = true;
  } else {
    boardDevice[index].libertyNumber = libertyNumber;
  }

  __syncthreads();
  __threadfence_block();

  if (hasStoneRemoved){
  
    globalLiberty[index] = 0;
  
    __syncthreads();
    __threadfence_block();
  
    updateLiberty(boardDevice, index, globalLiberty);
  
    __syncthreads();
    __threadfence_block();
  
    libertyNumber = globalLiberty[boardDevice[index].groupID];
    boardDevice[index].libertyNumber = libertyNumber;
  }

//
//
//
//  if (boardDevice[index].pointGroup != NULL){
//    debugFlagDevice[index].changeFlag = boardDevice[index].pointGroup.numberOfLiberty; 
//    
//  }
//
//
//    debugFlagDevice[index].counter++;
//  }
//  

}

__device__
inline int inverseColor(int color){
  if (color == GO_BLACK){
    return GO_WHITE;
  }else if(color == GO_WHITE){
    return GO_BLACK;
  }
  return GO_EMPTY;
}

__global__
void updateLegleMove(BoardPoint *boardDevice, DebugFlag *debugFlagDevice, int color){
  int index = threadIdx.y*boardSize + threadIdx.x;

  if (boardDevice[index].color == GO_EMPTY){
    int totalLiberty = 0;
    
    if (boardDevice[index - 1].color == color){
      totalLiberty = totalLiberty + boardDevice[index - 1].libertyNumber - 1;
    }else if(boardDevice[index - 1].color == GO_EMPTY){
      totalLiberty++;
    }

    if (boardDevice[index + 1].color == color){
      totalLiberty = totalLiberty + boardDevice[index + 1].libertyNumber - 1;
    }else if(boardDevice[index + 1].color == GO_EMPTY){
      totalLiberty++;
    }

    if (boardDevice[index - boardSize].color == color){
      totalLiberty = totalLiberty + boardDevice[index - boardSize].libertyNumber - 1;
    }else if(boardDevice[index - boardSize].color == GO_EMPTY){
      totalLiberty++;
    }

    if (boardDevice[index + boardSize].color == color){
      totalLiberty = totalLiberty + boardDevice[index + boardSize].libertyNumber - 1;
    }else if(boardDevice[index + boardSize].color == GO_EMPTY){
      totalLiberty++;
    }

    debugFlagDevice[index].libertyCount = totalLiberty;

    if (totalLiberty > 0){
      if (color == GO_BLACK){
        boardDevice[index].isBlackLegal = true;
      }else if (color == GO_WHITE){
        boardDevice[index].isWhiteLegal = true;
      }
    }else{
      if (color == GO_BLACK){
        boardDevice[index].isBlackLegal = false;
      }else if (color == GO_WHITE){
        boardDevice[index].isWhiteLegal = false;
      }
 
    }
    
  } else {
      if (color == GO_BLACK){
        boardDevice[index].isBlackLegal = false;
      }else if (color == GO_WHITE){
        boardDevice[index].isWhiteLegal = false;
      }
  }
     
}
 
int main()
{
  BoardPoint boardHost[totalSize];
  BoardPoint *boardDevice;
  DebugFlag debugFlagHost[totalSize];
  DebugFlag *debugFlagDevice;

  const int valueSizeDevice = totalSize*sizeof(BoardPoint);
  const int debugFlagSize = totalSize*sizeof(DebugFlag);

  hipMalloc( (void**)&boardDevice, valueSizeDevice );
  hipMalloc( (void**)&debugFlagDevice, debugFlagSize );

  
  struct timeval start_tv;
  gettimeofday(&start_tv,NULL);
  
  
  dim3 threadShape( boardSize, boardSize );
  int numberOfBlock = 1;

  initBoard<<<numberOfBlock, threadShape>>>(boardDevice);
  
//  for (int i=0; i<19; i++){
//    playBoard<<<numberOfBlock, threadShape>>>(boardDevice, globalFlag, i, i, 2);
//  }

  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 10, 10, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 10, 11, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 10, 12, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 11, 10, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 12, 10, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 13, 10, 1);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 13, 9, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 13, 11, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 14, 10, 2);

  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 1, 1, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 2, 1, 1);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 1, 2, 1);

  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 5, 10, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 5, 11, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 5, 12, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 6, 10, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 7, 10, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 8, 10, 1);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 8, 9, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 8, 11, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 9, 10, 2);

  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 19, 19, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 18, 19, 1);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 19, 18, 1);

  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 10, 4, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 10, 5, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 10, 6, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 11, 4, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 12, 4, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 13, 4, 1);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 13, 3, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 13, 5, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 14, 4, 2);

  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 17, 16, 2);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 17, 17, 1);
  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 15, 12, 1);

  //updateLegleMove<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, GO_BLACK);
  //updateLegleMove<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, GO_WHITE);

  hipDeviceSynchronize();

  hipMemcpy( boardHost, boardDevice, valueSizeDevice, hipMemcpyDeviceToHost );
  hipMemcpy( debugFlagHost, debugFlagDevice, debugFlagSize, hipMemcpyDeviceToHost );


  hipFree( boardDevice );
  hipFree( debugFlagDevice );
  
  hipDeviceSynchronize();

  struct timeval end_tv;
  gettimeofday(&end_tv,NULL);
 
  for (int i=boardSize-1; i>=0; i--){
    for (int j=0; j<boardSize; j++){
      int index = i*boardSize + j;
      if (boardHost[index].color == 0){
        printf(".");
      }else if (boardHost[index].color == GO_BLACK){
        printf("o");
      }else if (boardHost[index].color == GO_WHITE){
        printf("x");
      }else if (boardHost[index].color == GO_BORDER){
        printf("H");
      }
    }
    printf("\n");
   
  }

//  for (int i=boardSize-1; i>=0; i--){
//    for (int j=0; j<boardSize; j++){
//      int index = i*boardSize + j;
////      if (boardHost[index].color == GO_BLACK || boardHost[index].color == GO_WHITE){
//        printf("%d, %d | ", boardHost[index].groupID, boardHost[index].libertyNumber);
////      } else if (boardHost[index].color == GO_EMPTY) {
////        printf("   ,   | ");
////      }
//    }
//    printf("\n");
//   
//  }

  for (int i=boardSize-1; i>=0; i--){
    for (int j=0; j<boardSize; j++){
      int index = i*boardSize + j;
      if (boardHost[index].color == GO_BORDER){
        printf("H");
      }else{
        if (boardHost[index].isBlackLegal){
          printf("o");
        }else {
          printf(".");
        }
      }
    }

    printf("        ");

    for (int j=0; j<boardSize; j++){
      int index = i*boardSize + j;
      if (boardHost[index].color == GO_BORDER){
        printf("H");
      }else{
        if (boardHost[index].isWhiteLegal){
          printf("x");
        }else {
          printf(".");
        }
      }
    }
    
    printf("\n");
   
  }



//  for (int i=boardSize-1; i>=0; i--){
//    for (int j=0; j<boardSize; j++){
//      int index = i*boardSize + j;
//      printf("%d | ", debugFlagHost[index].libertyCount);
//      }
//    printf("\n");
//   
//  }


  printf("\n");

  if(end_tv.tv_usec >= start_tv.tv_usec){
    printf("time %lu:%lu\n",end_tv.tv_sec - start_tv.tv_sec,  end_tv.tv_usec - start_tv.tv_usec);
  }else{
    printf("time %lu:%lu\n",end_tv.tv_sec - start_tv.tv_sec - 1,  1000000 - start_tv.tv_usec + end_tv.tv_usec);
  }

  
  return EXIT_SUCCESS;
  
}
