#include "hip/hip_runtime.h"
#include <stdio.h>
 
const int N = 16; 
 
__global__ 
void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}

__global__ 
void SumMultiple(int inputArray[N][N], int outputArray[N][N]){
  int i = threadIdx.x;
  int j = threadIdx.y;
  
  outputArray[i][j] = 0;

  if (i > 0 && i < N-1){
    if (j > 0 && j < N-1){
      outputArray[i][j] = outputArray[i][j] + inputArray[i-1][j-1];
      outputArray[i][j] = outputArray[i][j] + inputArray[i][j-1];
      outputArray[i][j] = outputArray[i][j] + inputArray[i][j-1];
      outputArray[i][j] = outputArray[i][j] + inputArray[i+1][j-1];

      outputArray[i][j] = outputArray[i][j] + inputArray[i-1][j];
      outputArray[i][j] = outputArray[i][j] + inputArray[i][j];
      outputArray[i][j] = outputArray[i][j] + inputArray[i][j];
      outputArray[i][j] = outputArray[i][j] + inputArray[i+1][j];

      outputArray[i][j] = outputArray[i][j] + inputArray[i-1][j+1];
      outputArray[i][j] = outputArray[i][j] + inputArray[i][j+1];
      outputArray[i][j] = outputArray[i][j] + inputArray[i][j+1];
      outputArray[i][j] = outputArray[i][j] + inputArray[i+1][j+1];

    }
    
  } 

}
 
int main()
{
  
  int inputArray[N][N];
  int outputArray[N][N];

  for (int i=0; i<N; i++){
    for (int j=0; j<N; j++){
      inputArray[i][j] = i*N+j;
      outputArray[i][j] = 0;
    }
  }

  const int arraySize = N*N*sizeof(int);

  int* inputDevice;
  int* outputDevice;

  size_t inputSizeT;
  size_t outputSizeT;

	hipMallocPitch( (void**)&inputDevice, &inputSizeT, N*sizeof(int), N ); 
  hipMallocPitch( (void**)&outputDevice, &outputSizeT, N*sizeof(int), N );

	hipMemcpy( inputDevice, inputArray, arraySize, hipMemcpyHostToDevice ); 
  hipMemcpy( outputDevice, outputArray, arraySize, hipMemcpyHostToDevice);
	
	int numberOfBlock = 1;
	dim3 threadPerBlock( N, N );

	SumMultiple<<<numberOfBlock, threadPerBlock>>>(inputDevice, outputDevice);
  
	hipMemcpy( outputArray, outputDevice, arraySize, hipMemcpyDeviceToHost ); 
	hipFree( inputDevice );
	hipFree( outputDevice );
	
	
  for (int i=0; i<N; i++){
    for (int j=0; j<N; j++){
      inputArray[i][j] = i*N+j;
      printf("%d ", outputArray[i][j]); 
    }
    printf("/n");
  }

	return EXIT_SUCCESS;
}
