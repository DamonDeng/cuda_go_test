
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <time.h>

 
const int N = 16; 
const int blocksize = 16; 
 
__global__ 
void hello(int *a, int *b) 
{
  for (int i=0; i<10000000; i++){
  	a[threadIdx.x] += b[threadIdx.x];

  }
}
 
int main()
{
	int a[N]; // = {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1};
	int b[N]; // = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 
  for (int i=0; i<N; i++){
    a[i] = 1;
    b[i] = 1;
  }

  struct timeval start_tv;
  gettimeofday(&start_tv,NULL);
  //printf("time %u:%u\n",tv.tv_sec,tv.tv_usec);

  //time_t t = time(NULL);
  //struct tm tm = *localtime(&t);

  //printf("year: %d \n", tm.tm_year);

  //std::time_t startTime = std::time(nullptr);
  //time_t startTime = time(NULL);
  //time(&startTime);
  
	int *ad;
	int *bd;
	const int csize = N*sizeof(int);
	const int isize = N*sizeof(int);

  //for (int j=0; j<10000; j++){
	  hipMalloc( (void**)&ad, csize ); 
	  hipMalloc( (void**)&bd, isize ); 
	  hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	  hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	  
	  dim3 dimBlock( blocksize, 1 );
	  dim3 dimGrid( 1, 1 );
	  hello<<<dimGrid, dimBlock>>>(ad, bd);
	  hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	  hipFree( ad );
	  hipFree( bd );


  //}
  
  hipDeviceSynchronize();

  //time_t endTime;
  //time(&endTime);
  struct timeval end_tv;
  gettimeofday(&end_tv,NULL);
 
  for (int i=0; i<N; i++){
    printf("%d ", a[i]);
	   
  }

  printf("\n");

  //printf("start time: %f \n", startTime);
  //printf("end time: %f \n", endTime);
  //printf("time used: %f \n", endTime-startTime);
  if(end_tv.tv_usec >= start_tv.tv_usec){
    printf("time %u:%u\n",end_tv.tv_sec - start_tv.tv_sec,  end_tv.tv_usec - start_tv.tv_usec);
  }else{
    printf("time %u:%u\n",end_tv.tv_sec - start_tv.tv_sec,  1000000 - start_tv.tv_usec + end_tv.tv_usec);
  }

  
  return EXIT_SUCCESS;
}
