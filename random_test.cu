#include <stdio.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

const int boardSize = 21; 
const int totalSize = boardSize * boardSize;

struct BoardPoint{
  int color;
  int groupID;
  int libertyNumber;
  bool isBlackLegal;
  bool isWhiteLegal;
};


__global__
void randomInit(hiprandState *state, long randSeed){
  int index = threadIdx.y*boardSize + threadIdx.x;
  //curandState state;
//  long seed = 123456;
  hiprand_init(randSeed, index, 0, &state[index]);
//  boardPoint[index].color = curand(&state);

}


__global__
void randomTest(BoardPoint *boardPoint, hiprandState *state){
  int index = threadIdx.y*boardSize + threadIdx.x;
//  curandState state;
//  long seed = 123456;
//  curand_init(seed, index, 0, &state);
  boardPoint[index].color = (hiprand(&state[index])>>16)%361;

}

int main()
{
  BoardPoint boardHost[totalSize];
  BoardPoint *boardDevice;
  hiprandState *stateDevice;
//  DebugFlag debugFlagHost[totalSize];
//  DebugFlag *debugFlagDevice;
//
  const int valueSizeDevice = totalSize*sizeof(BoardPoint);
//  const int debugFlagSize = totalSize*sizeof(DebugFlag);
//
  hipMalloc( (void**)&boardDevice, valueSizeDevice );
  hipMalloc( (void**)&stateDevice, valueSizeDevice );
//  cudaMalloc( (void**)&debugFlagDevice, debugFlagSize );
//
//  
  dim3 threadShape( boardSize, boardSize );
  int numberOfBlock = 1;

  srand((unsigned int)time(NULL));

  randomInit<<<numberOfBlock, threadShape>>>(stateDevice, rand());

  struct timeval start_tv;
  gettimeofday(&start_tv,NULL);
  
  
  randomTest<<<numberOfBlock, threadShape>>>(boardDevice, stateDevice);
  
//  for (int i=0; i<19; i++){
//    playBoard<<<numberOfBlock, threadShape>>>(boardDevice, globalFlag, i, i, 2);
//  }
  hipDeviceSynchronize();

  hipMemcpy( boardHost, boardDevice, valueSizeDevice, hipMemcpyDeviceToHost );
//  cudaMemcpy( debugFlagHost, debugFlagDevice, debugFlagSize, cudaMemcpyDeviceToHost );
//
//
  hipFree( boardDevice );
//  cudaFree( debugFlagDevice );
//  
  hipDeviceSynchronize();
//
  struct timeval end_tv;
  gettimeofday(&end_tv,NULL);

  for (int i=boardSize-1; i>=0; i--){
    for (int j=0; j<boardSize; j++){
      int index = i*boardSize + j;
      printf("%d| ",boardHost[index].color);
    }
    printf("\n");
   
  }

 

  if(end_tv.tv_usec >= start_tv.tv_usec){
    printf("time %lu:%lu\n",end_tv.tv_sec - start_tv.tv_sec,  end_tv.tv_usec - start_tv.tv_usec);
  }else{
    printf("time %lu:%lu\n",end_tv.tv_sec - start_tv.tv_sec - 1,  1000000 - start_tv.tv_usec + end_tv.tv_usec);
  }

  
  return EXIT_SUCCESS;
  
}
